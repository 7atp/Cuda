#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
 
#include <omp.h>

void fillVector(int * v, size_t n);
void addVector(int * a, int *b, int *c, size_t n);
void printVector(int * v, size_t n);
void serial();
void cudacode();
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size,int n);

int main()
{
	const int n = 2;
	hipEvent_t start; hipEventCreate(&start);
	hipEvent_t stop; hipEventCreate(&stop);
	const int vectorSize = n*1024;
	int a[vectorSize], b[vectorSize], c[vectorSize];

	fillVector(a, vectorSize);
	fillVector(b, vectorSize);
	float msecTotal = 0.0f;
	for (int i = 0; i < 50; i++) {
		hipEventRecord(start, NULL);
		
		addWithCuda(c, a, b, vectorSize,n);
		hipEventRecord(stop, NULL);
		hipEventSynchronize(stop);
		float msecTotal1 = 0.0f;
		hipEventElapsedTime(&msecTotal1, start, stop);
		msecTotal += msecTotal1;
	}
	printf("this is the mean elapsed time %f", msecTotal/50.0);
	/*double starttime, elapsedtime;

	starttime = omp_get_wtime();

	addVector(a, b, c, vectorSize);


	elapsedtime = omp_get_wtime() - starttime;
	printf("this is the mean elapsed time for serial %f", elapsedtime ); */


	printVector(c, vectorSize);

	return EXIT_SUCCESS;
}

__global__ void addKernel(int *c, const int *a, const int *b, const int *n) {
	int i = threadIdx.x;
	for (int j = 0; j < n[0]; j++) {
		c[i+j] = a[i+j] + b[i+j];
	}
}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size,int n){

	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	int *dev_n = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0); 
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); }
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); }
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); }

	cudaStatus = hipMalloc((void**)&dev_n,  sizeof(int));
	if (cudaStatus != hipSuccess) { printf("hipMalloc failed!"); }

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice); 
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); }
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); }

	cudaStatus = hipMemcpy(dev_n, &n,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); }

	addKernel <<<1, 1024 >>> (dev_c, dev_a, dev_b,dev_n);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { printf("hipMemcpy failed!"); }
	hipFree(dev_c); hipFree(dev_a); hipFree(dev_b); hipFree(dev_n);

	return cudaStatus;



}



// Fills a vector with data
void fillVector(int * v, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		v[i] = i;
	}
}

// Adds two vectors
void addVector(int * a, int *b, int *c, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}

// Prints a vector to the stdout.
void printVector(int * v, size_t n) {
	int i;
	printf("[-] Vector elements: ");
	for (i = 0; i < n; i++) {
		printf("%d, ", v[i]);
	}
	printf("\b\b  \n");
}
